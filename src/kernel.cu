#include "hip/hip_runtime.h"
#include <kernel.h>
#include <iostream>
#include <stdio.h>


/*
 * Naive Kernel
 */

__global__ void naiveKernelTE(hipfftDoubleComplex* E, hipfftDoubleComplex* N,
	hipfftDoubleComplex* KX, hipfftDoubleComplex* S, double* X, 
	double k0, double dz, int nx, int i) {
	
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(j<nx) {

		// load constants and variables
		hipfftDoubleComplex li = make_hipDoubleComplex(0,1);
		hipfftDoubleComplex result = make_hipDoubleComplex(0,0);
		
		hipfftDoubleComplex n = N[i*nx+j];
		hipfftDoubleComplex next_n = N[(i+1)*nx+j];

		// preliminary computing
		double nk0 = n.x * k0;
		double kk0 = n.y * k0;
		double next_nk0 = next_n.x * k0;

		hipfftDoubleComplex KZ, next_KZ, F, tmp;

		// loop over frequencies
		for(int kj=0; kj<nx; kj++) {

			// compute KZ and next_KZ
			tmp = KX[kj];	KZ = cuCsqrt(nk0*nk0-tmp*tmp);
			tmp = KX[kj];	next_KZ = cuCsqrt(next_nk0*next_nk0-tmp*tmp);

			if( KZ.y < 1e-6 && KZ.x!=0 ) {	// exclude evanescent waves

				// Fresnel coefficient
				F = KZ*2/(KZ+next_KZ);
				
				// compute E
				int sign = (kj%2)? -1:1;
				result = result + 
						( F * (S[kj]/nx) * (sign*std::exp(-kk0*dz)) * 
						cuCexp(li*(KX[kj]*X[j]+KZ*dz)) );
			}
		}
		E[(i+1)*nx+j] = result;
	}
}

__global__ void naiveKernelTM(hipfftDoubleComplex* E, hipfftDoubleComplex* N,
	hipfftDoubleComplex* KX, hipfftDoubleComplex* S, double* X, 
	double k0, double dz, int nx, int i) {
	
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(j<nx) {

		// load constants and variables
		hipfftDoubleComplex li = make_hipDoubleComplex(0,1);
		hipfftDoubleComplex result = make_hipDoubleComplex(0,0);
		
		hipfftDoubleComplex n = N[i*nx+j];
		hipfftDoubleComplex next_n = N[(i+1)*nx+j];

		// preliminary computing
		double nk0 = n.x * k0;
		double kk0 = n.y * k0;
		double next_nk0 = next_n.x * k0;

		hipfftDoubleComplex KZ, next_KZ, F, tmp;

		// loop over frequencies
		for(int kj=0; kj<nx; kj++) {

			// compute KZ and next_KZ
			tmp = KX[kj];	KZ = cuCsqrt(nk0*nk0-tmp*tmp);
			tmp = KX[kj];	next_KZ = cuCsqrt(next_nk0*next_nk0-tmp*tmp);

			if( KZ.y < 1e-6 && KZ.x!=0 ) {	// exclude evanescent waves

				// Fresnel coefficient
				F = n*next_n*KZ*2 / (next_n*next_n*KZ+n*n*next_KZ);
				
				// compute E
				int sign = (kj%2)? -1:1;
				result = result + 
						( F * (S[kj]/nx) * (sign*std::exp(-kk0*dz)) * 
						cuCexp(li*(KX[kj]*X[j]+KZ*dz)) );
			}
		}
		E[(i+1)*nx+j] = result;
	}
}

void naiveKernelWrapper(hipfftDoubleComplex* E, hipfftDoubleComplex* N, 
	hipfftDoubleComplex* KX, double* X, double k0, double dz, int nz, int nx,
	int blockSize, int fresnel) {
	
	hipfftHandle plan;
	hipfftPlan1d(&plan, nx, HIPFFT_Z2Z, 1);

	hipfftDoubleComplex* S;
	hipMalloc(&S, nx*sizeof(hipfftDoubleComplex));

	dim3 block_dim(blockSize, 1, 1);
	dim3 grid_dim(ceil(static_cast<double>(nx)/static_cast<double>(blockSize)), 1, 1);

//	std::cout << grid_dim.x << "*" << blockSize << "\t" << std::flush;

	switch(fresnel) {
	case 0:
		for(int i=0; i<nz-1; i++) {
			// fftt
			hipfftExecZ2Z(plan, E+i*nx, S, HIPFFT_FORWARD);
			// compute E
			naiveKernelTE<<<grid_dim, block_dim>>>(E, N, KX, S, X, k0, dz, nx, i);
		}
		break;
	
	case 1:
		for(int i=0; i<nz-1; i++) {
			// fftt
			hipfftExecZ2Z(plan, E+i*nx, S, HIPFFT_FORWARD);
			// compute E
			naiveKernelTM<<<grid_dim, block_dim>>>(E, N, KX, S, X, k0, dz, nx, i);
		}
		break;
		
	default:
		std::cout << "Error - Unknown Fresnel Coefficient" << std::endl;
		exit(-1);
	}
}



/*
 * Shared Memory Kernel
 */

__global__ void shMemKernelTE(hipfftDoubleComplex* E, hipfftDoubleComplex* N,
	hipfftDoubleComplex* KX, hipfftDoubleComplex* S, double* X, 
	double k0, double dz, int nx, int i) {
	
	// initialite constants and variables
	extern __shared__ hipfftDoubleComplex shMem[];
	hipfftDoubleComplex* kx = shMem;
	hipfftDoubleComplex* s = &shMem[blockDim.x];

	int nb = nx/blockDim.x;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	hipfftDoubleComplex li, n, next_n;
	hipfftDoubleComplex tmp, result;
	hipfftDoubleComplex KZ, next_KZ, F;
	double nk0, kk0, next_nk0;

	// preliminary computing
	if(j<nx) {
		li = make_hipDoubleComplex(0,1);
		result = make_hipDoubleComplex(0,0);
		
		n = N[i*nx+j];
		next_n = N[(i+1)*nx+j];
		
		nk0 = n.x * k0;
		kk0 = n.y * k0;
		next_nk0 = next_n.x * k0;
	}

	for(int l=0; l<nb; l++) {
		// collaboratively load kx
		if(l*blockDim.x + threadIdx.x < nx ) {
			kx[threadIdx.x] = KX[l*blockDim.x + threadIdx.x];
			s[threadIdx.x] = S[l*blockDim.x + threadIdx.x];
		}
		__syncthreads();

		if(j<nx) {
			for(int ll=0; ll<blockDim.x; ll++) {

				// compute KZ and next_KZ
				tmp = kx[ll];	KZ = cuCsqrt(nk0*nk0-tmp*tmp);
				tmp = kx[ll];	next_KZ = cuCsqrt(next_nk0*next_nk0-tmp*tmp);

				if( KZ.y < 1e-6 && KZ.x!=0 ) {	// exclude evanescent waves

					// Fresnel coefficient
					F = KZ*2/(KZ+next_KZ);

					// compute E
					int sign = ( (l*blockDim.x+ll)%2 )? -1:1;
					result = result + 
						( F * (s[ll]/nx) * (sign*std::exp(-kk0*dz)) * 
						cuCexp(li*(kx[ll]*X[j]+KZ*dz)) );
				}	
			}
		}
		__syncthreads();
	}
	E[(i+1)*nx+j] = result;
}

__global__ void shMemKernelTM(hipfftDoubleComplex* E, hipfftDoubleComplex* N,
	hipfftDoubleComplex* KX, hipfftDoubleComplex* S, double* X, 
	double k0, double dz, int nx, int i) {
	
	// initialite constants and variables
	extern __shared__ hipfftDoubleComplex shMem[];
	hipfftDoubleComplex* kx = shMem;
	hipfftDoubleComplex* s = &shMem[blockDim.x];

	int nb = nx/blockDim.x;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	hipfftDoubleComplex li;
	hipfftDoubleComplex tmp, result, n, next_n;
	hipfftDoubleComplex KZ, next_KZ, F;
	double nk0, kk0, next_nk0;

	// preliminary computing
	if(j<nx) {
		li = make_hipDoubleComplex(0,1);
		result = make_hipDoubleComplex(0,0);
		
		n = N[i*nx+j];
		next_n = N[(i+1)*nx+j];
		
		nk0 = n.x * k0;
		kk0 = n.y * k0;
		next_nk0 = next_n.x * k0;
	}

	for(int l=0; l<nb; l++) {
		// collaboratively load kx
		if(l*blockDim.x + threadIdx.x < nx ) {
			kx[threadIdx.x] = KX[l*blockDim.x + threadIdx.x];
			s[threadIdx.x] = S[l*blockDim.x + threadIdx.x];
		}
		__syncthreads();

		if(j<nx) {
			for(int ll=0; ll<blockDim.x; ll++) {

				// compute KZ and next_KZ
				tmp = kx[ll];	KZ = cuCsqrt(nk0*nk0-tmp*tmp);
				tmp = kx[ll];	next_KZ = cuCsqrt(next_nk0*next_nk0-tmp*tmp);

				if( KZ.y < 1e-6 && KZ.x!=0 ) {	// exclude evanescent waves

					// Fresnel coefficient
					F = n*next_n*KZ*2 / (next_n*next_n*KZ+n*n*next_KZ);

					// compute E
					int sign = ( (l*blockDim.x+ll)%2 )? -1:1;
					result = result + 
						( F * (s[ll]/nx) * (sign*std::exp(-kk0*dz)) * 
						cuCexp(li*(kx[ll]*X[j]+KZ*dz)) );
				}	
			}
		}
		__syncthreads();
	}
	E[(i+1)*nx+j] = result;
}

void shMemKernelWrapper(hipfftDoubleComplex* E, hipfftDoubleComplex* N, 
	hipfftDoubleComplex* KX, double* X, double k0, double dz, int nz, int nx,
	int blockSize, int fresnel) {
	
	hipfftHandle plan;
	hipfftPlan1d(&plan, nx, HIPFFT_Z2Z, 1);

	hipfftDoubleComplex* S;
	hipMalloc(&S, nx*sizeof(hipfftDoubleComplex));

	dim3 block_dim(blockSize, 1, 1);
	dim3 grid_dim(ceil(static_cast<double>(nx)/static_cast<double>(blockSize)), 1, 1);
	int sharedMemorySize = 2 * blockSize * sizeof(hipfftDoubleComplex); 

//	std::cout << grid_dim.x << "*" << blockSize << " "
//		<< sharedMemorySize << "\t" << std::flush;

	switch(fresnel) {
	case 0:
		for(int i=0; i<nz-1; i++) {
			// fftt
			hipfftExecZ2Z(plan, E+i*nx, S, HIPFFT_FORWARD);
			// compute E
			shMemKernelTE<<<grid_dim, block_dim, sharedMemorySize>>>(
				E, N, KX, S, X, k0, dz, nx, i);
		}
		break;
	
	case 1:
		for(int i=0; i<nz-1; i++) {
			// fftt
			hipfftExecZ2Z(plan, E+i*nx, S, HIPFFT_FORWARD);
			// compute E
			shMemKernelTM<<<grid_dim, block_dim, sharedMemorySize>>>(
				E, N, KX, S, X, k0, dz, nx, i);
		}
		break;
		
	default:
		std::cout << "Error - Unknown Fresnel Coefficient " << fresnel << std::endl;
		exit(-1);
	}
}


/* complex math functions */
__host__ __device__ static __inline__ hipDoubleComplex hipCsub(double  x, hipDoubleComplex y) {
	return make_hipDoubleComplex(x - hipCreal(y), -hipCimag(y));
}

__host__ __device__ static __inline__ hipDoubleComplex hipCmul(hipDoubleComplex x, double y) {
	return make_hipDoubleComplex(hipCreal(x) * y, hipCimag(x) * y);
}

__host__ __device__ static __inline__ hipDoubleComplex hipCdiv(hipDoubleComplex x, double y) {
	return make_hipDoubleComplex(hipCreal(x) / y, hipCimag(x) / y);
}

__host__ __device__ static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex x) {
	double factor = std::exp(hipCreal(x));
	return make_hipDoubleComplex(factor * std::cos(hipCimag(x)), factor * std::sin(hipCimag(x)));
}

__host__ __device__ static __inline__ hipDoubleComplex cuCsqrt(hipDoubleComplex x) {
	if(x.x==0 && x.y==0) return make_hipDoubleComplex(0, 0);
	
	double radius = hipCabs(x);
	double cosA = x.x / radius;
	hipDoubleComplex out;
	out.x = std::sqrt(radius * (cosA + 1.0) / 2.0);
	out.y = std::sqrt(radius * (1.0 - cosA) / 2.0);
	// signbit should be false if x.y is negative
	if (signbit(x.y))
		out.y *= -1.0;

	return out;
}

/* overload */

__host__ __device__ static __inline__ hipDoubleComplex operator-(double x, hipDoubleComplex y) {
	return hipCsub(x,y);
}

__host__ __device__ static __inline__ hipDoubleComplex operator*(hipDoubleComplex x, double y) {
	return hipCmul(x,y);
}

__host__ __device__ static __inline__ hipDoubleComplex operator/(hipDoubleComplex x, double y) {
	return hipCdiv(x,y);
}

__host__ __device__ static __inline__ hipDoubleComplex operator+(hipDoubleComplex x, hipDoubleComplex y) {
	return hipCadd(x,y);
}

__host__ __device__ static __inline__ hipDoubleComplex operator*(hipDoubleComplex x, hipDoubleComplex y) {
	return hipCmul(x,y);
}

__host__ __device__ static __inline__ hipDoubleComplex operator/(hipDoubleComplex x, hipDoubleComplex y) {
	return hipCdiv(x,y);
}


